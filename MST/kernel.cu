
#include "hip/hip_runtime.h"
#include ""
#include <thrust/sort.h>
#include <thrust/iterator/zip_iterator.h>
#include <iostream>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <vector>
#include <thrust/remove.h>
#include <stdio.h>
const int v = 4;
using thrust::device_vector;

typedef struct {
	int src, dest, weight;
}borderland;
borderland* vert;

struct bordersort
{
	__host__ __device__
	bool operator()(borderland a, borderland b) {
		if (a.src == b.src) {
			if (a.dest == b.dest)
				return a.weight < b.weight;
			return a.dest < b.dest;
		}
		return a.src < b.src;
	}
};

struct borderselect
{
	__host__ __device__
	void operator()(borderland bdr) {
		int src = bdr.src;
		int dst = bdr.dest;
		int wgt = bdr.weight;
		
	}
};

int main()
{
	std::vector<borderland> tmp;
	hipMalloc((void**)&vert, sizeof(int) * v);
	tmp.push_back({ 0,1,10 });
	tmp.push_back({ 0,2,6 });
	tmp.push_back({ 0,2,10 });
	tmp.push_back({ 0,3,5 });
	tmp.push_back({ 2,3,4 });
	tmp.push_back({ 1,3,15 });
	device_vector<borderland> borders(tmp);
	thrust::sort(borders.begin(), borders.end(), bordersort());
	thrust::for_each(borders.begin(), borders.end(), borderselect());
	thrust::copy(borders.begin(), borders.end(), tmp.begin());
	for (auto i : tmp)
		std::cout << i.src << "\t" << i.dest << "\t" << i.weight << std::endl;
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
